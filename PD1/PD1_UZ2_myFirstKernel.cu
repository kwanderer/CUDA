#include "hip/hip_runtime.h"
// includes, system
//#include <time.h>
//#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

// Part 3 of 5: implement the kernel
__global__ void myFirstKernel(int *d_a)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_a[idx] = 1000 * blockIdx.x + threadIdx.x;

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    // pointer for host memory
    int* h_a;

    // pointer for device memory
    int* d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Part 1 of 5: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int*)malloc(memSize);
    hipMalloc(&d_a, memSize);

    // Part 2 of 5: configure and launch kernel
    dim3 dimGrid();
    dim3 dimBlock();
    myFirstKernel<<<numBlocks, numThreadsPerBlock >>>(d_a);
    //myFirstKernel<<<dimGrid(numBlocks), dimBlock(numThreadsPerBlock)>>>(d_a);
    //myFirstKernel<<<dimGrid(), dimBlock()>>>(d_a);

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");

    // Part 5 of 5: verify the data returned to the host is correct
    for (int i = 0; i < numBlocks; i++)
    {
        for (int j = 0; j < numThreadsPerBlock; j++)
        {
            assert(h_a[i * numThreadsPerBlock + j] == 1000 * i + j);
        }
    }

    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
