// includes, system
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include <iostream>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    // pointer and dimension for host memory
    int n, dimA;
    float* h_a;

    // pointers for device memory
    float* d_a, * d_b;

    // allocate and initialize host memory
    // Bonus: try using cudaMallocHost in place of malloc
    dimA = 8;
    h_a = (float*)malloc(dimA * sizeof(float));
    for (n = 0; n < dimA; n++)
    {
        h_a[n] = (float)n;
    }

    // Part 1 of 5: allocate device memory
    size_t memSize = dimA * sizeof(float);
    hipMalloc(&d_a, memSize);
    hipMalloc(&d_b, memSize);

    // Part 2 of 5: host to device memory copy
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);

    // Part 3 of 5: device to device memory copy
    hipMemcpy(d_b, d_a, memSize, hipMemcpyDeviceToDevice);

    // clear host memory
    for (n = 0; n < dimA; n++)
    {
        h_a[n] = 0.f;
    }

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls");

    // verify the data on the host is correct
    for (n = 0; n < dimA; n++)
    {
        assert(h_a[n] == (float)n);
    }

    // Part 5 of 5: free device memory pointers d_a and d_b
    hipFree(d_a);
    hipFree(d_b);

    // Check for any CUDA errors
    checkCUDAError("hipFree");

    // free host memory pointer h_a
    // Bonus: be sure to use cudaFreeHost for memory allocated with cudaMallocHost
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
